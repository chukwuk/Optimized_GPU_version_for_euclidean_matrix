#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <cstdlib>
#include <cmath>
#include <string>
#include <hip/hip_runtime.h>

#include "PrimAlgorithm.h"

using namespace std;

inline
hipError_t checkCudaErrors(hipError_t result, string functioncall = "")
{
//#if defined(DEBUG) || defined(_DEBUG)
  //fprintf(stderr, "CUDA Runtime Error: %d\n", result);
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error for this function call ( %s ) : %s\n", 
            functioncall.c_str(), hipGetErrorString(result));
    assert(result == hipSuccess);
  }
//#endif
  return result;
}

int
main( int argc, char* argv[ ] )
{ 
  //srand(time(0));
  unsigned long long int NUMDATA = 60000;
  const unsigned long long int bytes = NUMDATA * (long long int) sizeof(LocationPrim);
  const unsigned long long int bytes4euc = ( NUMDATA *  NUMDATA * (long long int)sizeof(float));
  fprintf (stderr, "Amount of data transfered to the device is %lld GB\n", bytes4euc/1000000000);
  //float time = 1.0;
  LocationPrim* locate = new LocationPrim[NUMDATA];
  for (int i = 0; i < NUMDATA; i++) {
      locate[i].x = rand() % 101;
      locate[i].y = rand() % 101;
      if (i == 0) {      
         fprintf (stderr, "%10.4f\n", locate[i].x);
         fprintf (stderr, "%10.4f\n", locate[i].y);     
      }
  }
  fprintf (stderr, "%10.4f\n", locate[NUMDATA-1].x);
  fprintf (stderr, "%10.4f\n", locate[NUMDATA-1].y);

  // Allocate memory on device
  float *distanceBtwAllLocation;
  LocationPrim *cordinateLocation;
  // Allocate memory on host
  float *HdistanceBtwAllLocation = new float[NUMDATA * NUMDATA];
   
  int BLOCKSIZE = 128;
  int NUMBLOCKS = (NUMDATA + BLOCKSIZE - 1)/BLOCKSIZE;
    
  fprintf (stderr, "NUMBER OF BLOCKS is %d\n", NUMBLOCKS);
  
  

  // Create CUDA events
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Record the start event
  hipEventRecord(start, 0); 

  hipError_t status;
  //allocate memory on the GPU device
  status = hipMalloc( (void **)(&distanceBtwAllLocation), bytes4euc);
  // checks for cuda errors  
  checkCudaErrors( status, "hipMalloc( (void **)(&distanceBtwAllLocation), bytes4euc)");
  
  // allocate memory on the GPU device
  status = hipMalloc( (void **)(&cordinateLocation), bytes);
  // checks for cuda errors
  checkCudaErrors( status, "hipMalloc( (void **)(&cordinateLocation), bytes)");

  // copy data from host memory to the device:

  status = hipMemcpy(cordinateLocation, locate, bytes, hipMemcpyHostToDevice );
  // checks for cuda errors
  checkCudaErrors( status,"hipMemcpy(cordinateLocation, locate, bytes, hipMemcpyHostToDevice )" );  
   
  // allocate number of threads in a block  
  dim3 threads(BLOCKSIZE, 1, 1 );

  // allocate number of blocks
  dim3 grid(NUMBLOCKS, 1, 1 );
  
  // call the kernel
  euclideanMatrix<<< grid, threads >>>( cordinateLocation, distanceBtwAllLocation,   NUMDATA);
  
  status = hipDeviceSynchronize( );
  
  checkCudaErrors( status,"euclideanMatrix<<< grid, threads >>>( cordinateLocation, distanceBtwAllLocation,   NUMDATA)");  
 
  status = hipGetLastError(); 
  
  checkCudaErrors( status,"hipGetLastError()");  

  // copy data device memory to host:
  hipMemcpy(HdistanceBtwAllLocation, distanceBtwAllLocation, bytes4euc, hipMemcpyDeviceToHost);  
  // checks for cuda errors
  checkCudaErrors( status );

  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
   
  // Calculate elapsed time
  float GpuTime = 0;
  hipEventElapsedTime(&GpuTime, start, stop); 
  
   
  printf("  GPU time: %f milliseconds\n", GpuTime);
  //printf("  Device to Host bandwidth (GB/s): %f\n", HdistanceBtwAllLocation[NUMDATA*(NUMDATA-1)] / time);
  //printf("  Device to Host bandwidth (GB/s): %f\n", HdistanceBtwAllLocation[NUMDATA-1] / time);

  // free device memory 
  hipFree( distanceBtwAllLocation );
  hipFree( cordinateLocation ); 
  
  // free host memory
  delete[] HdistanceBtwAllLocation;
  

   
  /* Running it on CPU************************************/  
  
  // Allocate memory on host
  float** AllLocationDistance = new float* [NUMDATA];
  
  for (int i = 0; i < NUMDATA; i++) {
     AllLocationDistance[i] = new float[NUMDATA];
  }
  
  // Record the start event
  hipEventRecord(start, 0); 
    
  for (int i = 0; i < NUMDATA; i++) {
     for (int j = i; j < NUMDATA; j++) {
         float x_co =  (locate[i].x - locate[j].x);
           float y_co =  (locate[i].y - locate[j].y);
           float pow_xco = powf(x_co, 2.0);
           float pow_yco = powf(y_co, 2.0);
       	   float pow_plus = powf((pow_yco+pow_xco), 0.5);
           AllLocationDistance[i][j] = pow_plus;
           if (i < j) { 
              AllLocationDistance[j][i] = pow_plus;
           }
     }
  }
    
  // Record the stop event
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop); 
     
  // Calculate elapsed time
  float CpuTime = 0;
  hipEventElapsedTime(&CpuTime, start, stop); 


  // Clean up
  hipEventDestroy(start);
  hipEventDestroy(stop);
  
  printf("  CPU time: %f milliseconds\n", CpuTime);
  //printf("  Device to Host bandwidth (GB/s): %f\n", bytes4euc*1e-9/time);
  //double check = 99999.000*100000.000;
  //printf("  Device to Host bandwidth (GB/s): %f\n", check);
  
  // free host memory
  delete[] locate;
   
    
  for (int i = 0; i < NUMDATA; i++) {
     delete[] AllLocationDistance[i]; 
  }
  
  delete[] AllLocationDistance; 

  return 0;

};	
